
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

const int matrixSize = 512;

__global__ void kernel_mm(int* matA, int* matB, int* product, int matrixSize){
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int row = threadIndex / matrixSize;
    int column = threadIndex % matrixSize;
    for (int k = 0; k < matrixSize; k++){
        product[row * matrixSize + column] += matA[row * matrixSize + k] * matB[k * matrixSize + column];
    }
}

int* allocateMatrix(int n){
    return (int*)malloc(n * n * 4);
}

void fillMatrix(int n, int* mat){
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            mat[i * n + j] = rand() % 5;
        }
    }
}

void printMatrix(int n, int* mat){
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            printf("%d ", mat[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int* hostMat1 = allocateMatrix(matrixSize);
    fillMatrix(matrixSize, hostMat1);
    int* hostMat2 = allocateMatrix(matrixSize);
    fillMatrix(matrixSize, hostMat2);
    int* hostMat3 = allocateMatrix(matrixSize);
    for (int i = 0; i < matrixSize; i++){
        for (int j = 0; j < matrixSize; j++){
            hostMat3[i * matrixSize + j] = 0;
        }
    }

    //printMatrix(matrixSize, hostMat1);
    //printMatrix(matrixSize, hostMat2);
    //printMatrix(matrixSize, hostMat3);

    size_t bytes = matrixSize * matrixSize * sizeof(int);
    int *deviceMat1, *deviceMat2, *deviceMat3;
    hipMalloc(&deviceMat1, bytes);
    hipMalloc(&deviceMat2, bytes);
    hipMalloc(&deviceMat3, bytes);
    hipMemcpy(deviceMat1, hostMat1, bytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceMat2, hostMat2, bytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceMat3, hostMat3, bytes, hipMemcpyHostToDevice);

    const int threads = 512;
    const int blocks = (matrixSize * matrixSize) / threads;
    dim3 gridSize(blocks, 1, 1);
    dim3 blockSize(threads, 1, 1);

    hipEventRecord(start);
    kernel_mm<<<gridSize, blockSize>>>(deviceMat1, deviceMat2, deviceMat3, matrixSize);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipMemcpy(hostMat3, deviceMat3, bytes, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel duration: %3.6fms\n", milliseconds);
    //printMatrix(matrixSize, hostMat3);

    hipFree(deviceMat1);
    hipFree(deviceMat2);
    hipFree(deviceMat3);
    free(hostMat1);
    free(hostMat2);
    free(hostMat3);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
