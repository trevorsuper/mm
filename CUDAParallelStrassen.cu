
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int n = 512;

void printMatrix(int n, int** mat){
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            printf("%d ", mat[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}

int* allocateMatrix(int n){
    return (int*)malloc(n * n * sizeof(int));
}

int** allocateMatrix2D(int n){
    int* data = (int*)malloc(n * n * sizeof(int));
    int** array = (int**)malloc(n * sizeof(int*));
    for (int i = 0; i < n; i++){
        array[i] = &(data[n * i]);
    }
    return array;
}

void fillMatrix2D(int n, int** &mat){
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            mat[i][j] = rand() % 5;
        }
    }
}

int** getSlice(int n, int** mat, int offseti, int offsetj){
    int m = n / 2;
    int** slice = allocateMatrix2D(m);
    for (int i = 0; i < m; i++){
        for (int j = 0; j < m; j++){
            slice[i][j] = mat[offseti + i][offsetj + j];
        }
    }
    return slice;
}

int** addMatrices(int n, int** mat1, int** mat2, bool add){
    int** result = allocateMatrix2D(n);
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            if (add)
                result[i][j] = mat1[i][j] + mat2[i][j];
            else
                result[i][j] = mat1[i][j] - mat2[i][j];
        }
    }
    return result;
}

int** combineMatrices(int m, int** c11, int** c12, int** c21, int** c22){
    int n = 2 * m;
    int** result = allocateMatrix2D(n);
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            if (i < m && j < m)
                result[i][j] = c11[i][j];
            else if (i < m)
                result[i][j] = c12[i][j - m];
            else if (j < m)
                result[i][j] = c21[i - m][j];
            else
                result[i][j] = c22[i - m][j - m];
        }
    }
    return result;
}

void freeMatrix(int n, int* mat){
    free(mat);
}

void freeMatrix2D(int n, int** mat){
    free(mat[0]);
    free(mat);
}

int** naive(int n, int** mat1, int** mat2){
    int** product = allocateMatrix2D(n);
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            product[i][j] = 0;
            for (int k = 0; k < n; k++){
                product[i][j] += mat1[i][k] * mat2[k][j];
            }
        }
    }
    return product;
}

__global__ void kernel_mm(int* matA, int* matB, int* product, int matrixSize){
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int row = threadIndex / matrixSize;
    int column = threadIndex % matrixSize;
    for (int k = 0; k < matrixSize; k++) {
        product[row * matrixSize + column] += matA[row * matrixSize + k] * matB[k * matrixSize + column];
    }
}

int** cudaNaive(int n, int** mat1, int** mat2){
    int* hostMat1 = allocateMatrix(n);
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            hostMat1[i * n + j] = mat1[i][j];
        }
    }

    int* hostMat2 = allocateMatrix(n);
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            hostMat2[i * n + j] = mat2[i][j];
        }
    }

    int* hostProduct = allocateMatrix(n);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            hostProduct[i * n + j] = 0;
        }
    }

    size_t bytes = n * n * sizeof(int);
    int *deviceMat1, *deviceMat2, *deviceProduct;
    hipMalloc(&deviceMat1, bytes);
    hipMalloc(&deviceMat2, bytes);
    hipMalloc(&deviceProduct, bytes);
    hipMemcpy(deviceMat1, hostMat1, bytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceMat2, hostMat2, bytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceProduct, hostProduct, bytes, hipMemcpyHostToDevice);

    const int threads = 512;
    const int blocks = (n * n) / threads;
    dim3 gridSize(blocks, 1, 1);
    dim3 blockSize(threads, 1, 1);

    kernel_mm<<<gridSize, blockSize>>>(deviceMat1, deviceMat2, deviceProduct, n);
    hipDeviceSynchronize();

    hipMemcpy(hostProduct, deviceProduct, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(deviceMat1);
    hipFree(deviceMat2);
    hipFree(deviceProduct);
    freeMatrix(n, hostMat1);
    freeMatrix(n, hostMat2);

    int** product = allocateMatrix2D(n);
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            product[i][j] = hostProduct[i * n + j];
        }
    }
    return product;
}

int** strassen(int n, int** mat1, int** mat2){
    int m = n / 2;

    int** a = getSlice(n, mat1, 0, 0);
    int** b = getSlice(n, mat1, 0, m);
    int** c = getSlice(n, mat1, m, 0);
    int** d = getSlice(n, mat1, m, m);
    int** e = getSlice(n, mat2, 0, 0);
    int** f = getSlice(n, mat2, 0, m);
    int** g = getSlice(n, mat2, m, 0);
    int** h = getSlice(n, mat2, m, m);

    int** bds = addMatrices(m, b, d, false);
    int** gha = addMatrices(m, g, h, true);
    int** s1 = cudaNaive(m, bds, gha);
    freeMatrix2D(m, bds);
    freeMatrix2D(m, gha);

    int** ada = addMatrices(m, a, d, true);
    int** eha = addMatrices(m, e, h, true);
    int** s2 = cudaNaive(m, ada, eha);
    freeMatrix2D(m, ada);
    freeMatrix2D(m, eha);

    int** acs = addMatrices(m, a, c, false);
    int** efa = addMatrices(m, e, f, true);
    int** s3 = cudaNaive(m, acs, efa);
    freeMatrix2D(m, acs);
    freeMatrix2D(m, efa);

    int** aba = addMatrices(m, a, b, true);
    int** s4 = cudaNaive(m, aba, h);
    freeMatrix2D(m, aba);
    freeMatrix2D(m, b);

    int** fhs = addMatrices(m, f, h, false);
    int** s5 = cudaNaive(m, a, fhs);
    freeMatrix2D(m, fhs);
    freeMatrix2D(m, a);
    freeMatrix2D(m, f);
    freeMatrix2D(m, h);

    int** ges = addMatrices(m, g, e, false);
    int** s6 = cudaNaive(m, d, ges);
    freeMatrix2D(m, ges);
    freeMatrix2D(m, g);

    int** cda = addMatrices(m, c, d, true);
    int** s7 = cudaNaive(m, cda, e);
    freeMatrix2D(m, cda);
    freeMatrix2D(m, c);
    freeMatrix2D(m, d);
    freeMatrix2D(m, e);

    int** s1s2a = addMatrices(m, s1, s2, true);
    int** s6s4s = addMatrices(m, s6, s4, false);
    int** c11 = addMatrices(m, s1s2a, s6s4s, true);
    freeMatrix2D(m, s1s2a);
    freeMatrix2D(m, s6s4s);
    freeMatrix2D(m, s1);

    int** c12 = addMatrices(m, s4, s5, true);
    freeMatrix2D(m, s4);

    int** c21 = addMatrices(m, s6, s7, true);
    freeMatrix2D(m, s6);

    int** s2s3s = addMatrices(m, s2, s3, false);
    int** s5s7s = addMatrices(m, s5, s7, false);
    int** c22 = addMatrices(m, s2s3s, s5s7s, true);
    freeMatrix2D(m, s2s3s);
    freeMatrix2D(m, s5s7s);
    freeMatrix2D(m, s2);
    freeMatrix2D(m, s3);
    freeMatrix2D(m, s5);
    freeMatrix2D(m, s7);

    int** product = combineMatrices(m, c11, c12, c21, c22);

    freeMatrix2D(m, c11);
    freeMatrix2D(m, c12);
    freeMatrix2D(m, c21);
    freeMatrix2D(m, c22);

    return product;
}

int main(){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int** mat1 = allocateMatrix2D(n);
    fillMatrix2D(n, mat1);
    int** mat2 = allocateMatrix2D(n);
    fillMatrix2D(n, mat2);
    hipEventRecord(start);
    int** mat3 = strassen(n, mat1, mat2);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel duration: %3.6fms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //printMatrix(n, mat1);
    //printMatrix(n, mat2);
    //printMatrix(n, mat3);
    freeMatrix2D(n, mat1);
    freeMatrix2D(n, mat2);
    freeMatrix2D(n, mat3);
    return 0;
}
